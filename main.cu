#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>

#include<thrust/scan.h>
#include<thrust/functional.h>

#include<iostream>

int add_(int a, int b)
{
	return a+b;
}

__device__ int log_plus(int a, int b)
{
        return a+b;
}

__device__ int segscan_warp(int* ptr, bool* hd, int idx) {
        const unsigned int lane = idx & 31;
        if (lane >= 1) {
                ptr[idx] = hd[idx] ? ptr[idx] : log_plus(ptr[idx - 1] , ptr[idx]);
                hd[idx] = hd[idx - 1] | hd[idx]; }
        if (lane >= 2) {
                ptr[idx] = hd[idx] ? ptr[idx] : log_plus(ptr[idx - 2] , ptr[idx]);
                hd[idx] = hd[idx - 2] | hd[idx]; }
        if (lane >= 4) {
                ptr[idx] = hd[idx] ? ptr[idx] : log_plus(ptr[idx - 4] , ptr[idx]);
                hd[idx] = hd[idx - 4] | hd[idx]; }
        if (lane >= 8) {
                ptr[idx] = hd[idx] ? ptr[idx] : log_plus(ptr[idx - 8] , ptr[idx]);
                hd[idx] = hd[idx - 8] | hd[idx]; }
        if (lane >= 16) {
                ptr[idx] = hd[idx] ? ptr[idx] : log_plus(ptr[idx - 16] , ptr[idx]);
                hd[idx] = hd[idx - 16] | hd[idx];
        }
	return ptr[idx];
}

__device__ void segscan_block(int* ptr, bool* hd, int idx)
{
	unsigned int warpid = idx >> 5;
        unsigned int warp_first = warpid << 5;
        unsigned int warp_last = warp_first + 31;
        // Step 1a:
        // Before overwriting the input head flags, record whether // this warp begins with an "open" segment.
        bool warp_is_open = (hd[warp_first] == 0);
        __syncthreads ();
        // Step 1b:
        // Intra-warp segmented scan in each warp.
        int val = segscan_warp(ptr, hd, idx);
        // Step 2a:
        // Since ptr[] contains *inclusive* results, irrespective of Kind, // the last value is the correct partial result.
        int warp_total = ptr[warp_last];
        // Step 2b:
        // warp_flag is the OR-reduction of the flags in a warp and is
        // computed indirectly from the mindex values in hd[].
        // will_accumulate indicates that a thread will only accumulate a
        // partial result in Step 4 if there is no segment boundary to its left. 
        bool warp_flag = hd[warp_last]!=0 || !warp_is_open;
        bool will_accumulate = warp_is_open && hd[idx]==0;
        __syncthreads ();
        // Step 2c: The last thread in each warp writes partial results
        if( idx == warp_last ) {
                ptr[warpid] = warp_total;
                hd[warpid] = warp_flag;
        }
        __syncthreads ();
        // Step 3: One warp scans the per-warp results
        if( warpid == 0 ) segscan_warp(ptr, hd, idx);
        __syncthreads ();
        // Step 4: Accumulate results from
        if( warpid != 0 && will_accumulate)
                val = log_plus(ptr[warpid -1], val);
        __syncthreads ();
        ptr[idx] = val;
        __syncthreads ();
}

__global__ void kernel_1(int* array, int size, bool* key)
{
	int idx=threadIdx.x;
	int stt=blockIdx.x;
	segscan_block(array+stt*1838, key+stt*1838, idx);
	__syncthreads();
	if(threadIdx.x==0&&key[1024+stt*1838]==0)
	{
		key[1024+stt*1838]=1;
		array[1024+stt*1838]+=array[1023+stt*1838];
	}
	__syncthreads();
	if(threadIdx.x+1024<size)
	{
		segscan_block(array+1024+stt*1838, key+1024+stt*1838, idx);
	}
}
int main()
{
	int* data;
	bool* keys;
	int* vals;
	int* thrust_keys;
	int n=10;
	int array_size=1838*n;
	int num_of_rule=1838;
	printf("%f\n",-std::numeric_limits<float>::max());
	hipMallocManaged(&thrust_keys, array_size*sizeof(int));
	hipMallocManaged(&data, array_size*sizeof(int));
	hipMallocManaged(&keys, array_size*sizeof(bool));
	hipMallocManaged(&vals, array_size*sizeof(int));
	
	for(int i=0;i<num_of_rule;i++)
	{
		for(int j=0;j<n;j++)
		{
			data[i+j*num_of_rule] = 1;
			thrust_keys[i+j*num_of_rule]=i/10;
			if(i%10==0) keys[i+j*num_of_rule]=1;
			else keys[i+j*num_of_rule]=0;
		}
	}

	thrust::equal_to<int> binary_pred;
	for(int i=0;i<n;i++)
	thrust::inclusive_scan_by_key(thrust_keys+i*num_of_rule, thrust_keys +i*num_of_rule+ 1838, data+i*num_of_rule, vals+i*num_of_rule, binary_pred, add_);
	dim3 grid(n);
	dim3 block(1024);
	dim3 block2(1838-1024);
	kernel_1<<< grid, block >>>(data, 1838, keys);
	hipDeviceSynchronize();
	for(int i=0;i<1838*n;i++) {
		if(vals[i]!=data[i]) printf("%d ", i);
		if((vals[i]-1)%10!=(i%1838)%10) printf("%d vs %d\n", vals[i], i);
	}
	printf("\n");
	return 0;
}
